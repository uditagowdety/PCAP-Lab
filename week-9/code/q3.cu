#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

__global__ void modifymatrix(int* matrix, int* outputmatrix, int m, int n) {
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    int col = threadIdx.y + blockIdx.y * blockDim.y;

    if (row < m && col < n) {
        // If it's a border element, leave it the same
        if (row == 0 || row == m - 1 || col == 0 || col == n - 1) {
            outputmatrix[row * n + col] = matrix[row * n + col];
        } else {
            // Non-border elements, replace with 1's complement in binary
            int value = matrix[row * n + col];
            outputmatrix[row * n + col] = ~value; // 1's complement
        }
    }
}

int main() {
    int m, n;
    printf("Enter matrix dimensions (m x n): ");
    scanf("%d %d", &m, &n);

    int* matrix = (int*)malloc(m * n * sizeof(int));
    int* outputmatrix = (int*)malloc(m * n * sizeof(int));

    printf("Enter matrix elements:\n");
    for (int i = 0; i < m * n; i++) {
        scanf("%d", &matrix[i]);
    }

    // Device pointers for matrix
    int* d_matrix, *d_outputmatrix;
    hipMalloc((void**)&d_matrix, m * n * sizeof(int));
    hipMalloc((void**)&d_outputmatrix, m * n * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_matrix, matrix, m * n * sizeof(int), hipMemcpyHostToDevice);

    // Define thread block and grid dimensions
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((m + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                   (n + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Launch the CUDA kernel
    modifymatrix<<<numBlocks, threadsPerBlock>>>(d_matrix, d_outputmatrix, m, n);

    // Check for kernel execution errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        return -1;
    }

    // Copy result from device to host
    hipMemcpy(outputmatrix, d_outputmatrix, m * n * sizeof(int), hipMemcpyDeviceToHost);

    // Display the modified matrix
    printf("\nModified Matrix:\n");
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            printf("%d ", outputmatrix[i * n + j]);
        }
        printf("\n");
    }

    // Free memory
    free(matrix);
    free(outputmatrix);
    hipFree(d_matrix);
    hipFree(d_outputmatrix);

    return 0;
}
