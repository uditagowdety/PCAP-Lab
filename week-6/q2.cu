
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void selection_sort(int *arr, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        for (int i = idx; i < n - 1; i++) {
            int min_idx = i;
            for (int j = i + 1; j < n; j++) {
                if (arr[j] < arr[min_idx]) {
                    min_idx = j;
                }
            }
            if (min_idx != i) {
                int temp = arr[i];
                arr[i] = arr[min_idx];
                arr[min_idx] = temp;
            }
        }
    }
}

void launch_selection_sort(int *arr, int n) {
    int *d_arr;
    size_t size = n * sizeof(int);
    hipMalloc((void**)&d_arr, size);
    hipMemcpy(d_arr, arr, size, hipMemcpyHostToDevice);
    
    int block_size = 256;
    int num_blocks = (n + block_size - 1) / block_size;
    selection_sort<<<num_blocks, block_size>>>(d_arr, n);
    
    hipMemcpy(arr, d_arr, size, hipMemcpyDeviceToHost);
    hipFree(d_arr);
}

int main() {
    int arr[] = {64, 25, 12, 22, 11};
    int n = sizeof(arr) / sizeof(arr[0]);

    launch_selection_sort(arr, n);

    for (int i = 0; i < n; i++) {
        printf("%d ", arr[i]);
    }
    printf("\n");
    return 0;
}