#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>

__global__ void count_word(char* sentence, char* word, int* count, int s_len, int w_len) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i <= s_len - w_len) {
        bool match = true;
        for (int j = 0; j < w_len && match; j++)
            if (sentence[i + j] != word[j]) match = false;
        if (match) atomicAdd(count, 1);
    }
}

int main() {
    char sentence[] = "this is a test sentence for testing", word[] = "test";
    int count = 0, s_len = strlen(sentence), w_len = strlen(word);
    char *d_sentence, *d_word;
    int *d_count;

    hipMalloc(&d_sentence, s_len + 1);
    hipMalloc(&d_word, w_len + 1);
    hipMalloc(&d_count, sizeof(int));

    hipMemcpy(d_sentence, sentence, s_len + 1, hipMemcpyHostToDevice);
    hipMemcpy(d_word, word, w_len + 1, hipMemcpyHostToDevice);
    hipMemcpy(d_count, &count, sizeof(int), hipMemcpyHostToDevice);

    count_word<<<(s_len / 256) + 1, 256>>>(d_sentence, d_word, d_count, s_len, w_len);
    hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);

    printf("word '%s' appears %d times\n", word, count);

    hipFree(d_sentence); hipFree(d_word); hipFree(d_count);
    return 0;
}
