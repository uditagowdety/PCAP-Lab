#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>

__global__ void generate_string(char* s, char* rs, int s_len) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < s_len) {
        for(int i = 0; i <= index; i++) rs[index * s_len + i] = s[i];
    }
}

int main() {
    char s[] = "PCAP";
    int s_len = strlen(s), rs_len = s_len * s_len;
    char *d_s, *d_rs, rs[rs_len];

    hipMalloc(&d_s, s_len * sizeof(char));
    hipMalloc(&d_rs, rs_len * sizeof(char));
    hipMemcpy(d_s, s, s_len * sizeof(char), hipMemcpyHostToDevice);

    generate_string<<<(s_len / 256) + 1, 256>>>(d_s, d_rs, s_len);
    hipMemcpy(rs, d_rs, rs_len * sizeof(char), hipMemcpyDeviceToHost);

    for(int i = 0; i < rs_len; i++) if(rs[i]) printf("%c", rs[i]);
    printf("\n");

    hipFree(d_s); hipFree(d_rs);
    return 0;
}
