#include<stdio.h>
#include "hip/hip_runtime.h"

__global__ void add(int* a, int* b, int* c){
    *c=*a+*b;
    printf("inside device: %d + %d = %d\n",*a, *b, *c);
}

int main(){
    int a,b,c;
    int size=sizeof(int);
    a=3;
    b=5;

    int *d_a, *d_b, *d_c;

    hipMalloc((void**)&d_a,size);
    hipMalloc((void**)&d_b,size);
    hipMalloc((void**)&d_c,size);

    hipMemcpy(d_a,&a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,&b,size,hipMemcpyHostToDevice);

    add<<<1,1>>>(d_a,d_b,d_c);

    hipMemcpy(&c,d_c,size,hipMemcpyDeviceToHost);

    printf("result from device: %d\n",c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}