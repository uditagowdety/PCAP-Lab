#include<stdio.h>
#include "hip/hip_runtime.h"

__global__ void element_add(int *a, int*b, int* c){
    int index=threadIdx.x;
    c[index]=a[index]+b[index];
}

int main(){
    int arr1[]={1,2,3,4,5};
    int arr2[]={6,7,8,9,10};
    int result[5];
    int size=sizeof(int)*5;

    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a,size);
    hipMalloc((void**)&d_b,size);
    hipMalloc((void**)&d_c,size);

    hipMemcpy(d_a,arr1,size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,arr2,size,hipMemcpyHostToDevice);

    element_add<<<1,5>>>(d_a,d_b,d_c);

    hipMemcpy(result,d_c,size,hipMemcpyDeviceToHost);

    printf("result array:  ");
    for (int i=0;i<5;i++) printf("%d ",result[i]);
    printf("\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}